 // Vector addition: C = A + B.
// compile with the following command:
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O2 -m64 -o vecAdd vecAdd.cu


// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float* h_D;
float* d_A;   // device vectors
float* d_B;
float* d_C;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void MatAdd(const float* A, const float* B, float* C, int N)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < N && j < N){
	    int index = i*N +j;
       	    C[index] = A[index] + B[index];
    }
    __syncthreads();
}

// Host code

int main( )
{

    int gid;   // GPU_ID

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    printf("Enter the GPU_ID: ");
    scanf("%d",&gid);
    printf("%d\n", gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Matrix Addition: C = A + B\n");
    int mem = 1024*1024*1024;     // Giga    
    int N;

    printf("Enter the size of the matrix: ");
    scanf("%d",&N);        
    printf("%d\n",N);        
    if( N*N > mem ) {     // each real number (float) takes 4 bytes
      printf("The size of these 3 vectors cannot be fitted into 6 Gbyte\n");
      exit(2);
    }
    long size = N * N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize the input vectors with random numbers

    RandomInit(h_A, N*N);
    RandomInit(h_B, N*N);

    // Set the sizes of threads and blocks


    int threadsPerBlock;
loop:
    printf("Enter the number of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if( threadsPerBlock > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      goto loop;
    }
    dim3 block(threadsPerBlock, threadsPerBlock);
    dim3 numBlocks((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    //int blocksPerGrid = (N + threadsPerBlock - 1)/threadsPerBlock;
    printf("The number of blocks is %d\n", numBlocks);
   
    
    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    MatAdd<<<numBlocks, block>>>(d_A, d_B, d_C, N);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",3*N/(1000000.0*gputime));
    
    // Copy result from device memory to host memory
    // h_C contains the result in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    h_D = (float*)malloc(size);       // to compute the reference solution
    for (int i = 0; i < N; ++i) 
        for (int j = 0; j < N; ++j){
		    int index = i*N + j;
		    h_D[index] = h_A[index] + h_B[index]; 
	    }
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",3*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result

    printf("Check result:\n");
    double sum=0; 
    double diff;
    for (int i = 0; i < N; ++i) {
         for (int j = 0; j < N; ++j){
		    int index = i*N + j;
		    diff = abs(h_D[index] - h_C[index]);
            sum += diff*diff; 
	    }
      
//      if(diff > 1.0e-15) { 
//        printf("i=%d, h_D=%15.10e, h_C=%15.10e \n", i, h_D[i], h_C[i]);
//      }
    }
    sum = sqrt(sum);
    printf("norm(h_C - h_D)=%20.15e\n\n",sum);

    hipDeviceReset();
}


// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for(int i = 0; i< n; i++)
        data[i] = rand() / (float)RAND_MAX;
}
